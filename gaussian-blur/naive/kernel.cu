#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
//#include "hip/device_functions.h"
#include ""
#include "hip/hip_runtime.h"
#include "book.h"
//#include "glut.h"
#include <GL/glut.h>
#include "stdio.h"
#include "iostream"
#include "math.h"

#define STB_IMAGE_IMPLEMENTATION
#include "../third-party/stb_image.h"

#define DIM 27     //GAUSSIAN FILTER DIMENSIONS MAX == 25 !!!
#define PI 3.1415926535897932385

//SPEEDING UP PURPOSES
__constant__ float matrix[DIM][DIM];

//DECLARED GLOBALLY
unsigned char *image_orig = NULL;
unsigned char *image_Gauss = NULL;
unsigned char *image = NULL;
int Width = NULL;
int Height = NULL;

/*
CREATE GAUSSIAN IMAGE FILTER MATRIX
SINGLE PRECISION IS QUITE ENOUGHT
*/
void fill(float rho,float *field){
	float e = 2.7182818284590452354f;
	for(int i =0;i<DIM;i++){
		for(int j=0;j<DIM;j++){
			int offset = DIM*i+j;
			//printf("   %d   \n",i);
			float x2 = pow((float)(j-(DIM/2)),2.0f);  //X RAISED TO POWER OF 2
			float y2 = pow((float)(i-(DIM/2)),2.0f);  //Y RAISED TO POWERR OF 2
			float rho2 = pow(rho,2.0f);               //RHO RAISED TO POWER OF 2
			float temp1 = -( (x2+y2)  / (2*rho2));    //EXPONENT OF E
			float tempx = pow(e,temp1);               //E RAISED TO POWERED OF TEMP1
			float base = 1/(2*PI*rho2);               //BASE
			float res = base*tempx;                   //RESULT OF GAUSSIAN FILTER FOR X & Y VALUES
			field[offset] = res;
		}
	}
};

//BEGIN OpenGL FUNCTIONS
static void Keyboard(unsigned char key, int x, int y) {
  switch (key) {
    case 27:
      exit(0);
  }
}

void myMenu(int menuItemID){

  switch(menuItemID) {
	case 1:
		image = image_orig;
		glutPostRedisplay();
		break;
	case 2:
		image = image_Gauss;  
		glutPostRedisplay();
		break;
    case 3:
      exit(0);
      break;
  }
}


void DrawImage( void ) {
   glClearColor( 0.0, 0.0, 0.0, 1.0 );
   glClear( GL_COLOR_BUFFER_BIT );
  if((image != NULL)) {
	glDrawPixels( Width, Height, GL_RGBA , GL_UNSIGNED_BYTE, image );
  }
  glutSwapBuffers();
 }
//END OpenGL FUNCTIONS


/*
MAIN GPU COMPUTING KERNEL
*/
__global__ void Gaussian(unsigned char *input,unsigned char *output,int *addittional)
{

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;
	int offset = x + y * blockDim.x * gridDim.x;
	int bloff = threadIdx.y * blockDim.x + threadIdx.x;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int bdx = blockDim.x; int bdy = blockDim.y;
	int width = addittional[0];
	int height = addittional[1];
	float conv_res = 0;

	
	//MAIN CONVOLUTION PART                                                             //////////////////// NEW //////////////////////////
	if (offset < (4*width * height)){
		int ii = 0;
		int jj = 0;
		
		for(int i = -DIM/2;i<(DIM/2);i++){
			for(int j = -DIM/2;j<(DIM/2);j++){
				int temp = 0; //DEFAULT VALUE WHEN OUT-OF-RANGE
				int new_x = x+j;
				int new_y = i+y;
				if((new_y<=height) && (new_y >= 0) && (new_x >=0 ) && (new_x <= width)){  //OUT-OF-RANGE?
					int new_offset = new_y*width + new_x;
					temp = input[4*new_offset+z];
				}

				conv_res +=temp*matrix[ii][jj];
				jj++;
			}
			jj = 0;
			ii++ ;
		}

		//RENEW USED VARIABLES AND SAVE
		tx = threadIdx.x; ty = threadIdx.y;
		output[4*offset+z] = (unsigned char) round(conv_res);
	}
	                                                                                   //////////////////// NEW //////////////////////////

}

int main(int argc,char **argv)
{
	hipEvent_t start,stop;
	float elapsedTime = 0;

	//CREATE FILTER MATRIX
	float rho = 8.0;
	float *host_matrix = (float*)malloc(DIM*DIM*sizeof(float));
	//float *dev_matrix;
	fill(rho,host_matrix);

	//IMAGE LOADING
	int chan = 4;
    int width,height,bits;  //x = WIDTH   y = HEIGHT n = BITS/Pix
    unsigned char *host_data = stbi_load("image.png", &width, &height, &bits, chan); //4 MEANS RGBA
	if(host_data == NULL) printf("Loading Failed \n");

	//DISPLAY IMAGE INFO
	Width = width;
	Height = height;
	printf("Image Size: %d Bits \n",width*height*bits);
	printf("WIDTH: %d Height: %d \n",width,height);

	//ALLOCATE
	unsigned char *host_res = (unsigned char*)malloc(width*height*chan*sizeof(char));
	int *host_addittional = (int*)malloc(3*sizeof(int));
	unsigned char *dev_input;
	unsigned char *dev_output;
	//float *dev_matrix;
	int *dev_addittional;
	host_addittional[0] = width;
	host_addittional[1] = height;
	host_addittional[2] = bits;


	//ALLOCATE FOR INPUT
	//HANDLE_ERROR(hipMalloc((void**)&dev_matrix,DIM*DIM*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_addittional,3*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_input,width*height*chan*sizeof(char)));
	HANDLE_ERROR(hipMalloc((void**)&dev_output,width*height*chan*sizeof(char)));

	//HANDLE_ERROR(hipMalloc((void**)&dev_matrix,DIM*DIM*sizeof(float)));
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start,0));


	//COPY INPUT
	HANDLE_ERROR(hipMemcpy(dev_input,host_data,chan*width*height*sizeof(char),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_addittional,host_addittional,3*sizeof(int),hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(dev_matrix,host_matrix,DIM*DIM*sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(matrix),host_matrix,DIM*DIM*sizeof(float)));


	//DEFINE DIMENSIONS
	//NOTE THAT WE ARE GOING TO LAUNCH 3D GRID (AS EACH Z.DIM for R,G,B,A)
	int th = 1024;
	dim3 BLOCKS_PER_GRID(width/th ,height,chan);
	dim3 THREADS_PER_BLOCK(th,1);

	//LAUNCH KERNEL
	Gaussian<<<BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(dev_input,dev_output,dev_addittional);

	//COPY BACK RESULTS
	HANDLE_ERROR(hipMemcpy(host_res,dev_output,chan*width*height*sizeof(char),hipMemcpyDeviceToHost));

	//GET AND DISPLAY CALCULATION TIME
	HANDLE_ERROR(hipEventRecord(stop,0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));
	printf("CUDA Gaussian Image Filter Completed in: %7.4f ms \n",elapsedTime);

	//DISPLAY IMAGE
	image_Gauss = host_res;
	image = host_data;
	image_orig = host_data;
	glutInitWindowSize(width, height);
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE |GLUT_RGBA);
    glutCreateWindow("Taylor Swift");
    glutKeyboardFunc(Keyboard);
	glutDisplayFunc(DrawImage);

	glutCreateMenu(myMenu);
	glutAddMenuEntry("Original", 1);
	glutAddMenuEntry("Gaussian", 2);
	glutAddMenuEntry("Exit", 3);
	glutAttachMenu(GLUT_RIGHT_BUTTON);

    glutMainLoop();

	//CLEAN MEMORY
	free(host_matrix);
	free(host_addittional);
	free(host_res);
	free(host_data);
	//hipFree(dev_matrix);
	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(dev_addittional);

    return 0;
   
}
